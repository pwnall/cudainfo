/** Prints CUDA GPU information in a machine-readable user-friendly format.
 *
 * The output can be read with a YAML parser, and is an array with one element
 * per CUDA GPU.
 *
 * Build with:
 *     nvcc -o cudainfo cudainfo.cu
 */
#include <stdio.h>

int main() {
  hipDeviceProp_t deviceProperties;
  hipError_t status;
  int deviceCount;

  if((status = hipGetDeviceCount(&deviceCount)) != hipSuccess) {
    fprintf(stderr, "CUDA error: %s\n", hipGetErrorString(status));
    return 1;
  }

  printf("---\n");
  for (int i = 0; i < deviceCount; ++i) {
    status = hipGetDeviceProperties(&deviceProperties, i);
    if(status != hipSuccess) {
      fprintf(stderr, "CUDA error: %s\n", hipGetErrorString(status));
      return 1;
    }

    printf("- name: \"%s\"\n", deviceProperties.name);
    printf("  compute_version: \"%d.%d\"\n",
           deviceProperties.major, deviceProperties.minor);
    printf("  pci_address: \"%02d:%02d\"\n",
           deviceProperties.pciBusID, deviceProperties.pciDeviceID);

    printf("  total_global_memory: %zu\n", deviceProperties.totalGlobalMem);
    printf("  total_constant_memory: %zu\n", deviceProperties.totalConstMem);
    printf("  shared_memory_per_block: %zu\n",
           deviceProperties.sharedMemPerBlock);
    printf("  max_malloc_pitch: %zu\n", deviceProperties.memPitch);
    printf("  texture_alignment: %zu\n", deviceProperties.textureAlignment);

    printf("  registers_per_block: %d\n", deviceProperties.regsPerBlock);
    printf("  max_threads_per_block: %d\n",
           deviceProperties.maxThreadsPerBlock);
    printf("  max_thread_block_dimension: [%d, %d, %d]\n",
           deviceProperties.maxThreadsDim[0], deviceProperties.maxThreadsDim[1],
           deviceProperties.maxThreadsDim[2]);
    printf("  max_grid_size: [%d, %d, %d]\n",
           deviceProperties.maxGridSize[0], deviceProperties.maxGridSize[1],
           deviceProperties.maxGridSize[2]);
    printf("  warp_size_threads: %d\n", deviceProperties.warpSize);

    printf("  multi_processor_count: %d\n",
           deviceProperties.multiProcessorCount);
    printf("  clock_rate_khz: %d\n", deviceProperties.clockRate);
    printf("  pci_bus_id: %d\n", deviceProperties.pciBusID);
    printf("  pci_device_id: %d\n", deviceProperties.pciDeviceID);
    printf("  compute_major: %d\n", deviceProperties.major);
    printf("  compute_minor: %d\n", deviceProperties.minor);

    printf("  integrated: %s\n",
           deviceProperties.integrated ? "true" : "false");
    printf("  supports_device_overlap: %s\n",
           deviceProperties.deviceOverlap ? "true" : "false");
    printf("  kernel_execution_timeout_enabled: %s\n",
           deviceProperties.kernelExecTimeoutEnabled ? "true" : "false");
    printf("  can_map_host_memory: %s\n",
           deviceProperties.canMapHostMemory ? "true" : "false");
    printf("  supports_concurrent_kernels: %s\n",
           deviceProperties.concurrentKernels ? "true" : "false");
    printf("  ecc_enabled: %s\n",
           deviceProperties.ECCEnabled ? "true" : "false");
    printf("  using_tcc_driver: %s\n",
           deviceProperties.tccDriver ? "true" : "false");

    const char* computeMode;
    switch (deviceProperties.computeMode) {
    case hipComputeModeDefault:
      computeMode = "default";
      break;
    case hipComputeModeExclusive:
      computeMode = "exclusive";
      break;
    case hipComputeModeProhibited:
      computeMode = "prohibited";
      break;
    default:
      computeMode = "unknown";
    }
    printf("  compute_mode: %s\n", computeMode);
  }

  return 0;
}
